#include "hip/hip_runtime.h"
// nvcc -std=c++17 -O3 -arch=sm_100a calculando_c.cu -lcuda

#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <time.h>
#include <iostream>



#include "ptx.cuh"
#include "utils.cuh"



using e8m0_t = uint8_t;
using fp8e4m3 = __hip_fp8_e4m3_fnuz;
typedef __hip_bfloat16 bf16;

constexpr int MXFP8_PREFETCH_BUFFERS_NUM = 0;
constexpr int K = 32;
constexpr int M = 64;
constexpr int BLOCK_SIZE_y = 32;
constexpr int BLOCK_SIZE_x = 64;
constexpr int NUM_THREADS = 64;
constexpr int buffers_num = 2;
constexpr int iterations = 2;
constexpr size_t MXFP8_CHUNKS_PER_BLOCK_Y = 1;
constexpr size_t MXFP8_CHUNKS_PER_BLOCK_X = 1;
constexpr size_t MXFP8_THREADS_PER_CHUNK = 64;



constexpr size_t MXFP8_CHUNK_DIM_Y = 64;
constexpr size_t MXFP8_ITERATIONS = MXFP8_CHUNK_DIM_Y / BLOCK_SIZE_y;  //   2 = 64 / 32


// Definir quantas iteração teremos por bloco e o formato de cada bloco (MAIOR), já temos os blocos menores 32 x 64

template <typename IType, typename OType>
__global__ void __launch_bounds__(MXFP8_THREADS_PER_CHUNK) cast_mxfp8_col_row_wise(
    const __grid_constant__ CUtensorMap tensor_map_input,
    const __grid_constant__ CUtensorMap tensor_map_output_rowwise,
    const __grid_constant__ CUtensorMap tensor_map_output_colwise,
    int width,
    int height,
    e8m0_t* scales_colwise,
    e8m0_t* scales_rowwise,
    bool is_col_wise,
    bool is_row_wise,
    bool output_colwise,
    bool output_rowwise
    ) {
        const int global_x = blockIdx.x * BLOCK_SIZE_x; 
        const int global_y = blockIdx.y * BLOCK_SIZE_x; // tem q ser 64 senao vai carregar os mesmos dados da next_iter
        const int tx = threadIdx.x;
        const int warp = threadIdx.x/32;
        __shared__ alignas(128) IType input[buffers_num][BLOCK_SIZE_y][BLOCK_SIZE_x];
        __shared__ alignas(128) OType output_colwise_sh[buffers_num][BLOCK_SIZE_y][BLOCK_SIZE_x];
        __shared__ alignas(128) OType output_rowwise_sh[buffers_num][BLOCK_SIZE_y][BLOCK_SIZE_x];
        #pragma nv_diag_suppress static_var_with_dynamic_init
        __shared__ alignas(8) uint64_t mbar[MXFP8_ITERATIONS];
        constexpr int shmem_buff_size = sizeof(input)/ buffers_num;
        const bool is_master_thread = (tx == 0);
        
        ptx::initialize_barriers<MXFP8_ITERATIONS, MXFP8_THREADS_PER_CHUNK>(mbar, is_master_thread);
        // usando TMA carregamos no buffer 0 do input
        ptx::copy_2d_to_shared(&input[0], &tensor_map_input, global_x,
            global_y, shmem_buff_size, &mbar[0],
            is_master_thread);

        int parity = 0;
        #pragma unroll
        for(int iter = 0; iter < iterations; iter++){
            const int next_iter = iter + 1;
            const int atual = iter % 2;
            if(next_iter < iterations){
                const int chunk_offset_x = global_x;
                const int chunk_offset_y = global_y + BLOCK_SIZE_y;
                ptx::copy_2d_to_shared(&input[next_iter % 2], &tensor_map_input, chunk_offset_x,
                    chunk_offset_y, shmem_buff_size, &mbar[next_iter],
                    is_master_thread);

            }
            ptx::fence_proxy_async_shared_cta();

            // Wait for the data to have arrived
            ptx::mbarrier_wait_parity(&mbar[iter], parity);

            if(is_col_wise){
                float max_val = 0.0;
                float elt;
                for(int i = 0; i < 32; i++){
                    elt = fabsf(__bfloat162float(input[atual][i][tx]));
                    max_val = elt > max_val ? elt : max_val;
                }
                const e8m0_t biased_exponent =
                float_to_e8m0(max_val * Quantized_Limits<OType>::max_norm_rcp);

                const float block_scale_inverse = exp2f_rcp(biased_exponent);

                for(int i = 0; i < 32; ++i) {
                    output_colwise_sh[atual][i][tx] =
                        static_cast<OType>(__bfloat162float(input[atual][i][tx]) * block_scale_inverse);
                }

                ptx::fence_proxy_async_shared_cta();
                __syncthreads();

                if(output_colwise && is_master_thread){ 
                    ptx::cp_async_bulk_tensor_2d_shared_to_global(reinterpret_cast<const uint64_t *>(&tensor_map_output_colwise), global_x, global_y + (iter * BLOCK_SIZE_y), reinterpret_cast<uint64_t *>(&output_colwise_sh[atual]));
                    

                    // Create a "bulk async-group" out of the previous bulk copy operation.
                    ptx::cp_async_bulk_commit_group();

                    // Wait for TMA transfer to have finished reading shared memory.
                    ptx::cp_async_bulk_wait_group_read<MXFP8_PREFETCH_BUFFERS_NUM>();
                }
                ptx::cp_async_bulk_wait_group_read<0>();
                __syncthreads();

                
                int buff = iter % 2;
                int scales_per_col = (BLOCK_SIZE_x * BLOCK_SIZE_x*(height/BLOCK_SIZE_x))/32;
                // suposta correção dos indices
                int dimensao_y = blockIdx.y * ((BLOCK_SIZE_y*BLOCK_SIZE_x)/K) + buff * (BLOCK_SIZE_x/K);
                int dimensao_x = blockIdx.x * scales_per_col + tx;
                scales_colwise[tx + (iter * BLOCK_SIZE_x)] = biased_exponent;
                
            }

            if(is_row_wise){
                float max_val = 0.0;
                int dim_x = (threadIdx.x % 2) * 32;
                int dim_y =  threadIdx.x/2;
                float elt;
                for(int i = 0; i < 32; i++){
                    elt = fabsf(__bfloat162float(input[atual][dim_y][dim_x + i]));
                    max_val = elt > max_val ? elt : max_val;
                    
                }
                max_val == 0.0 ? max_val = 1.0 : max_val;
                const e8m0_t biased_exponent =
                float_to_e8m0(max_val * Quantized_Limits<OType>::max_norm_rcp);
                const float block_scale_inverse = exp2f_rcp(biased_exponent);
                for(int j = 0; j < 32; j++){
                    output_rowwise_sh[atual][dim_y][dim_x + j] =
                        static_cast<OType>(__bfloat162float(input[atual][dim_y][dim_x + j]) * block_scale_inverse);
                }
                ptx::fence_proxy_async_shared_cta();
                __syncthreads();
                if(output_rowwise && is_master_thread){ 
                    ptx::cp_async_bulk_tensor_2d_shared_to_global(reinterpret_cast<const uint64_t *>(&tensor_map_output_rowwise), global_x, global_y + (iter * BLOCK_SIZE_y), reinterpret_cast<uint64_t *>(&output_rowwise_sh[atual]));
                
                
                    // Create a "bulk async-group" out of the previous bulk copy operation.
                    ptx::cp_async_bulk_commit_group();

                    // Wait for TMA transfer to have finished reading shared memory.
                    ptx::cp_async_bulk_wait_group_read<MXFP8_PREFETCH_BUFFERS_NUM>();
                }
                ptx::cp_async_bulk_wait_group_read<0>();
                __syncthreads();
                int block_tride_y = K / 32;
                // Suposta correção dos indices
                int buff = iter % 2;
                int dimensao_x = blockIdx.x * (BLOCK_SIZE_x/32) * M + (threadIdx.x % 2) * M + iter * BLOCK_SIZE_y;
                int dimensao_y = blockIdx.y * (BLOCK_SIZE_x * (width/32)) + threadIdx.x/2;
                scales_rowwise[tx + (iter * BLOCK_SIZE_x)] = biased_exponent; // block_stride_y seria qnt colunas input / 32 (primeira multiplicacao anda intra bloco[0,31], segunda anda extra bloco[0,qnt bloco x], dim_x fala se é o primeiro ou segundo scale intra bloco e global y anda extrabloco nas colunas)
                //blockIdx.x * (BLOCK_SIZE_x/32) * M + (threadIdx.x % 2) * M;
            }
            ptx::fence_proxy_async_shared_cta();
            __syncthreads();

            
            parity ^= 1;
        }
        
        ptx::destroy_barriers<MXFP8_ITERATIONS>(mbar, is_master_thread);

    
}

template <typename IType>
__host__ static inline CUtensorMap create_tensor_map(IType* gmem_ptr, int global_height, int global_width, 
    int smem_width, int smem_height) {
    CUtensorMap tensor_map;
    constexpr uint32_t rank = 2;
    uint64_t size[rank] = {(uint64_t)global_width, (uint64_t)global_height};
    uint64_t stride[rank - 1] = {(uint64_t)global_width * sizeof(IType)};
    uint32_t box_size[rank] = {(uint32_t)smem_width, (uint32_t)smem_height};
    void* gmem_address = (void*)gmem_ptr;
    uint32_t elem_stride[rank] = {1, 1};
    
    
    if constexpr (std::is_same_v<IType, __hip_bfloat16>){
    hipError_t result = cuTensorMapEncodeTiled(
        &tensor_map, CU_TENSOR_MAP_DATA_TYPE_BFLOAT16, rank, gmem_address, size,
        stride, box_size, elem_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE, CU_TENSOR_MAP_L2_PROMOTION_NONE, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);

    }
    else{
    hipError_t result = cuTensorMapEncodeTiled(
        &tensor_map, CU_TENSOR_MAP_DATA_TYPE_UINT8, rank, gmem_address, size,
        stride, box_size, elem_stride, CU_TENSOR_MAP_INTERLEAVE_NONE,
        CU_TENSOR_MAP_SWIZZLE_NONE, CU_TENSOR_MAP_L2_PROMOTION_NONE, CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE);
    }
    return tensor_map;
}

void quantBF16(bf16* data_ptr_Input, fp8e4m3* data_ptr_Output_colwise, fp8e4m3* data_ptr_Output_rowwise, uint8_t* data_ptr_scales_colwise, uint8_t* data_ptr_scales_rowwise) {
    int M = 64;
    int K = 64;

    // Tile sizes
    constexpr int BM = 64;
    constexpr int BK = 64;

    CUtensorMap tensor_map_Input;
    CUtensorMap tensor_map_Output_colwise;
    CUtensorMap tensor_map_Output_rowwise;
    int prev_m = 0;
    int prev_k = 0;

    // Check if we need to reallocate TMA maps
    if (M != prev_m) {
        // Allocate new TMA maps
        tensor_map_Input = create_tensor_map<bf16>(data_ptr_Input, K, M, BM, BK);
        tensor_map_Output_colwise = create_tensor_map<fp8e4m3>(data_ptr_Output_colwise, K, M, BM, BK);
        tensor_map_Output_rowwise = create_tensor_map<fp8e4m3>(data_ptr_Output_rowwise, K, M, BM, BK);
        
        prev_m = M;
        prev_k = K;
    }
    // Assert dimensions are correct
    assert(M == prev_m && K == prev_k);
    // Launch configuration
    dim3 grid((M/BM) * (K/BK));
    dim3 block(NUM_THREADS);

    // Launch kernel
    cast_mxfp8_col_row_wise<bf16, fp8e4m3><<<grid, block>>>(tensor_map_Input, tensor_map_Output_rowwise, tensor_map_Output_colwise, K, M, data_ptr_scales_colwise, data_ptr_scales_rowwise, 1, 1, 1, 1);

    hipDeviceSynchronize();



}

int main() {
    // Dimensões das matrizes (podem ser modificadas conforme necessário)
    const int ROWS = 64;
    const int COLS = 64;
    const int INPUT_SIZE = ROWS * COLS;
    
    // ==================== ALOCAÇÃO NO HOST ====================
    
    // Matriz Input (bf16) - será preenchida com valores
    bf16* h_input = new bf16[INPUT_SIZE];
    
    // Matrizes Output (E4M3) - serão zeradas
    fp8e4m3* h_output_colwise = new fp8e4m3[INPUT_SIZE];
    fp8e4m3* h_output_rowwise = new fp8e4m3[INPUT_SIZE];
    
    // Matrizes de scales (UINT8) - serão zeradas
    uint8_t* h_scales_colwise = new uint8_t[ROWS * (COLS / 32)];  // Uma escala por coluna
    uint8_t* h_scales_rowwise = new uint8_t[(ROWS / 32) * COLS];  // Uma escala por linha
    
    // ==================== PREENCHIMENTO DAS MATRIZES ====================
    
    for (int i = 0; i < INPUT_SIZE; i++) {
        h_input[i] = __float2bfloat16(i * 3.1421);
    }
    
    memset(h_output_colwise, 0, INPUT_SIZE * sizeof(fp8e4m3));
    memset(h_output_rowwise, 0, INPUT_SIZE * sizeof(fp8e4m3));
    
    memset(h_scales_colwise, 0, ROWS * (COLS / 32) * sizeof(uint8_t));
    memset(h_scales_rowwise, 0, (ROWS / 32) * COLS * sizeof(uint8_t));
    
    // ==================== ALOCAÇÃO NO DEVICE ====================
    
    
    // Ponteiros do device
    bf16* d_input;
    fp8e4m3* d_output_colwise;
    fp8e4m3* d_output_rowwise;
    uint8_t* d_scales_colwise;
    uint8_t* d_scales_rowwise;
    
    // Alocação das matrizes no device
    hipMalloc(&d_input, INPUT_SIZE * sizeof(bf16));
    hipMalloc(&d_output_colwise, INPUT_SIZE * sizeof(fp8e4m3));
    hipMalloc(&d_output_rowwise, INPUT_SIZE * sizeof(fp8e4m3));
    hipMalloc(&d_scales_colwise, ROWS * (COLS / 32) * sizeof(uint8_t));
    hipMalloc(&d_scales_rowwise, (ROWS / 32) * COLS * sizeof(uint8_t));
    
    // ==================== CÓPIA HOST -> DEVICE ====================
    
    
    hipMemcpy(d_input, h_input, INPUT_SIZE * sizeof(bf16), hipMemcpyHostToDevice);
    hipMemcpy(d_output_colwise, h_output_colwise, INPUT_SIZE * sizeof(fp8e4m3), hipMemcpyHostToDevice);
    hipMemcpy(d_output_rowwise, h_output_rowwise, INPUT_SIZE * sizeof(fp8e4m3), hipMemcpyHostToDevice);
    hipMemcpy(d_scales_colwise, h_scales_colwise, ROWS * (COLS / 32) * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(d_scales_rowwise, h_scales_rowwise, (ROWS / 32) * COLS * sizeof(uint8_t), hipMemcpyHostToDevice);


    quantBF16(d_input, d_output_colwise, d_output_rowwise, d_scales_colwise, d_scales_rowwise);

    // ==================== CÓPIA DEVICE -> HOST ====================
        
    hipMemcpy(h_input, d_input, INPUT_SIZE * sizeof(bf16), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_colwise, d_output_colwise, INPUT_SIZE * sizeof(fp8e4m3), hipMemcpyDeviceToHost);
    hipMemcpy(h_output_rowwise, d_output_rowwise, INPUT_SIZE * sizeof(fp8e4m3), hipMemcpyDeviceToHost);
    hipMemcpy(h_scales_colwise, d_scales_colwise, ROWS * (COLS / 32) * sizeof(uint8_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_scales_rowwise, d_scales_rowwise, (ROWS / 32) * COLS * sizeof(uint8_t), hipMemcpyDeviceToHost);
    
    
    // ==================== VERIFICAÇÃO DOS RESULTADOS ====================
    
    std::cout << "\n=== VERIFICAÇÃO DOS PRIMEIROS ELEMENTOS APÓS PROCESSAMENTO ===" << std::endl;
    
    // Printar primeiros elementos da matriz Input (bf16)
    std::cout << "\nPrimeiros 10 elementos da matriz Input (bf16):" << std::endl;
    for (int i = 0; i < 10 && i < INPUT_SIZE; i++) {
        std::cout << "Input[" << i << "] = " << __bfloat162float(h_input[i]) << std::endl;
    }
    
    // Printar primeiros elementos da matriz Output_colwise (E4M3)
    std::cout << "\nPrimeiros 10 elementos da matriz Output_colwise (E4M3):" << std::endl;
    for (int i = 0; i < 10 && i < INPUT_SIZE; i++) {
        // Conversão de E4M3 para float para visualização
        float val = __half2float((__half)h_output_colwise[i]);
        std::cout << "Output_colwise[" << i << "] = " << val << std::endl;
    }
    
    // Printar primeiros elementos da matriz Output_rowwise (E4M3)
    std::cout << "\nPrimeiros 10 elementos da matriz Output_rowwise (E4M3):" << std::endl;
    for (int i = 0; i < 10 && i < INPUT_SIZE; i++) {
        // Conversão de E4M3 para float para visualização
        float val = __half2float((__half)h_output_rowwise[i]);
        std::cout << "Output_rowwise[" << i << "] = " << val << std::endl;
    }
    
    // Printar primeiros elementos da matriz scales_colwise (UINT8)
    std::cout << "\nPrimeiros 10 elementos da matriz scales_colwise (UINT8):" << std::endl;
    for (int i = 0; i < 10 && i < COLS; i++) {
        std::cout << "scales_colwise[" << i << "] = " << (int)h_scales_colwise[i] << std::endl;
    }
    
    // Printar primeiros elementos da matriz scales_rowwise (UINT8)
    std::cout << "\nPrimeiros 10 elementos da matriz scales_rowwise (UINT8):" << std::endl;
    for (int i = 0; i < 10 && i < ROWS; i++) {
        std::cout << "scales_rowwise[" << i << "] = " << (int)h_scales_rowwise[i] << std::endl;
    }
    
    std::cout << "\n=== FIM DA VERIFICAÇÃO ===" << std::endl;


    // Liberar memória do device
    hipFree(d_input);
    hipFree(d_output_colwise);
    hipFree(d_output_rowwise);
    hipFree(d_scales_colwise);
    hipFree(d_scales_rowwise);
    
    
    // Liberar memória do host
    delete[] h_input;
    delete[] h_output_colwise;
    delete[] h_output_rowwise;
    delete[] h_scales_colwise;
    delete[] h_scales_rowwise;

    
    return 0;
}

